/*
 * Copyright (c) 2017-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "SingleHashGraph.cuh"

int main(int argc, char **argv) {

  int64_t countSize = 1L << 24;
  int64_t maxkey = 1L << 26;
  int64_t tableSize = maxkey;

  if (argc >= 2 && argc < 3) {
    std::cerr << "Please specify all arguments.\n";
    return 1;
  }

  if (argc >= 2) {
    uint32_t sizeExp = atoi(argv[1]);
    countSize = 1L << sizeExp;

    uint32_t keyExp = atoi(argv[2]);
    maxkey = 1L << keyExp;
  } 
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float buildTime = 0.0f; // milliseoncds

  rmm_mgpu_context_t context;

  std::cout << "before1" << std::endl;
  SingleHashGraph shg(countSize, maxkey, context, tableSize); 
  std::cout << "after1" << std::endl;

  hipEventRecord(start);

  shg.build(countSize, context, tableSize);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&buildTime, start, stop);

  std::cout << "single buildTable() time: " << (buildTime / 1000.0) << "\n"; // seconds
}
